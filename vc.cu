#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

__global__ void mul(int *a, int *b, int *c, int N) {
  // Global index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;


  for (int i = 0; i < N; i++) {
    int res = a[N*row+i] * b[N*i+col];
    c[N*row+col] += res;
  }
}

//Make  code  callable  from  Chapel
extern "C" {
  void mulCUDA(int *A, int *B, int *C, int start, int end, int GPUN) {

  double BLOCKSIZE = 32;
  int GRID = ceil(GPUN/BLOCKSIZE);

  int *da, *db, *dc; // Device  variables

  hipMalloc(&da, sizeof(int) * GPUN*GPUN);
  hipMalloc(&db, sizeof(int) * GPUN*GPUN);
  hipMalloc(&dc, sizeof(int) * GPUN*GPUN);

  //copy data to gpu
  hipMemcpy(da, A, sizeof(int) * GPUN*GPUN, hipMemcpyHostToDevice);
  hipMemcpy(db, B, sizeof(int) * GPUN*GPUN, hipMemcpyHostToDevice);

  //Kernel
  mul<<<grid, block>>>(da, db, dc, GPUN);

  // Copy back to host
  hipMemcpy(C, dc, sizeof(int)* GPUN*GPUN, hipMemcpyDeviceToHost);

  // Free gpu
  hipFree(da);
  hipFree(db);
  hipFree(dc);

  }
}
